#include "hip/hip_runtime.h"
#ifndef OUT_OF_BOUNDS_LABEL
#define OUT_OF_BOUNDS_LABEL -1
#endif

#ifndef BAD_TOPOLOGY_LABEL 
#define BAD_TOPOLOGY_LABEL -2
#endif

#ifndef NUM_OF_CHANNELS 
#define NUM_OF_CHANNELS 3
#endif


#ifndef USE_COUNTS
#define USE_COUNTS 1
#endif


#ifndef OUT_OF_BOUNDS_LABEL
#define OUT_OF_BOUNDS_LABEL -1
#endif

#define THREADS_PER_BLOCK 512


#include "update_seg.h"
#include "sp.h"

#include <stdio.h>
#ifndef WIN32
#include <unistd.h>
#endif


/*

__device__ static float atomicMaxFloat(float* address, float val)
{
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = ::atomicCAS(address_as_i, assumed,
            __float_as_int(::fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

*/

__device__ __forceinline__ float atomicMaxFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));

    return old;
}
__host__ void CudaFindBorderPixels(const int* seg, bool* border, const int nPixels, const int xdim, const int ydim, const int single_border){   
    int num_block = ceil( double(nPixels) / double(THREADS_PER_BLOCK) ); 
    dim3 ThreadPerBlock(THREADS_PER_BLOCK,1);
    dim3 BlockPerGrid(num_block,1);
    
    find_border_pixels<<<BlockPerGrid,ThreadPerBlock>>>(seg,border,nPixels, xdim, ydim, single_border);
}

__host__ void CudaFindBorderPixels_end(const int* seg, bool* border, const int nPixels, const int xdim, const int ydim, const int single_border){   
    int num_block = ceil( double(nPixels) / double(THREADS_PER_BLOCK) ); 
    dim3 ThreadPerBlock(THREADS_PER_BLOCK,1);
    dim3 BlockPerGrid(num_block,1);
    
    find_border_pixels_end<<<BlockPerGrid,ThreadPerBlock>>>(seg,border,nPixels, xdim, ydim, single_border);
}




__global__  void find_border_pixels(const int* seg, bool* border, const int nPixels, const int xdim, const int ydim, const int single_border){   
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  
    if (idx>=nPixels) return; 

    //border[idx]=0;  // init        
    int x = idx % xdim;
    int y = idx / xdim;

    int C =  __ldg(&seg[idx]); // center 
    int N,S,E,W; // north, south, east,west            
    /*N=S=W=E=OUT_OF_BOUNDS_LABEL; // init 
    if (y>1){
        N = seg[idx-xdim]; // above
    }          
    if (x>1){
        W = seg[idx-1];  // left
    }
    if (y<ydim-1){
        S = seg[idx+xdim]; // below
    }   
    if (x<xdim-1){
        E = seg[idx+1];  // right
    }       
   
    // If the nbr is different from the central pixel and is not out-of-bounds,
    // then it is a border pixel.
    if ( (N>=0 && C!=N) || (S>=0 && C!=S) || (E>=0 && C!=E) || (W>=0 && C!=W) ){

            border[idx]=1;  
        
    }
*/
    if ((y<1)||(x<1)||(y>=ydim-1)||(x>=xdim-1)) 
    {
        border[idx] = 1;
        return;
    }
    N = __ldg(&seg[idx-xdim]); // above
    W = __ldg(&seg[idx-1]);  // left
    S = __ldg(&seg[idx+xdim]); // below
    E = __ldg(&seg[idx+1]);  // right
           
   
    // If the nbr is different from the central pixel and is not out-of-bounds,
    // then it is a border pixel.
    if ((C!=N) || (C!=S) || (C!=E) || (C!=W) ){

            border[idx]=1;  
        
    }
    return;        
}


__global__  void find_border_pixels_end(const int* seg, bool* border, const int nPixels, const int xdim, const int ydim, const int single_border){   
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  
    if (idx>=nPixels) return; 

    border[idx]=0;  // init        
    
    int x = idx % xdim;
    int y = idx / xdim;

    int C = seg[idx]; // center 
    int N,S,E,W; // north, south, east,west            
    N=S=W=E=OUT_OF_BOUNDS_LABEL; // init 
    
    if (y>1){
        N = seg[idx-xdim]; // above
    }          
    if (x>1){
        W = seg[idx-1];  // left
    }
    if (y<ydim-1){
        S = seg[idx+xdim]; // below
    }   
    if (x<xdim-1){
        E = seg[idx+1];  // right
    }       
   
    // If the nbr is different from the central pixel and is not out-of-bounds,
    // then it is a border pixel.
    if ( (N>=0 && C!=N) || (S>=0 && C!=S) || (E>=0 && C!=E) || (W>=0 && C!=W) ){
            if (N>=0 && C>N) border[idx]=1; 
            if (S>=0 && C>S) border[idx]=1;
            if (E>=0 && C>E) border[idx]=1;
            if (W>=0 && C>W) border[idx]=1;
    }

    return;        
}



__global__   void cal_posterior( float* img, int* seg, bool* border, superpixel_params* sp_params, float3 J_i, float logdet_Sigma_i, float i_std, int s_std, int* changes, int nPts , int xdim,post_changes_helper* post_changes)
{
  /*  __shared__ float mu_i_x[720];
    __shared__ float mu_i_y[720];
    __shared__ float mu_i_z[720];
    __shared__ int mu_s_x[720];
    __shared__ int mu_s_y[720];
    if(threadIdx.x<720)
    {
        mu_i_x[720] = sp_params[threadIdx.x].mu_i.x;
        mu_i_y[720] = sp_params[threadIdx.x].mu_i.y;
        mu_i_z[720] = sp_params[threadIdx.x].mu_i.z;
        mu_s_x[720] = sp_params[threadIdx.x].mu_s.x;
        mu_s_y[720] = sp_params[threadIdx.x].mu_s.y;
    }
    __syncthreads();

    int idx_res = threadIdx.x + blockIdx.x*blockDim.x;
    int idx_inside = idx_res%4;
    int idx = idx_res/4;
	if (idx_res>=4*nPts)  return;
    int post = post_changes[idx].skip_post[idx_inside];
    int seg_idx = post_changes[idx].changes[idx_inside];
    if(post || ! seg_idx) return;

    float* imgC = img + idx * 3;
    int x = idx % xdim;  
    int y = idx / xdim;  
    float res = -1000; // some large negative number


    const float x0 = imgC[0]-mu_i_x[seg_idx];
    const float x1 = imgC[1]-mu_i_y[seg_idx];
    const float x2 = imgC[2]-mu_i_z[seg_idx];

    const int d0 = x - mu_s_x[seg_idx];
    const int d1 = y - mu_s_y[seg_idx];
    //color component
    const float J_i_x = J_i.x;
    const float J_i_y = J_i.y;
    const float J_i_z = J_i.z;
    const float sigma_s_x = sp_params[seg_idx].sigma_s.x;
    const float sigma_s_y = sp_params[seg_idx].sigma_s.y;
    const float sigma_s_z = sp_params[seg_idx].sigma_s.z;
    const float logdet_sigma_s = sp_params[seg_idx].logdet_Sigma_s;


    res = res - (x0*x0*J_i_x + x1*x1*J_i_y + x2*x2*J_i_z);   //res = -calc_squared_mahal_3d(imgC,mu_i,J_i);
    res = res -logdet_Sigma_i;
    //space component
    res = res - d0*d0*sigma_s_x;
    res = res - d1*d1*sigma_s_z;
    res = res -  2*d0*d1*sigma_s_y;            // res -= calc_squared_mahal_2d(pt,mu_s,J_s);
    res = res -  logdet_sigma_s;
    //post_changes[idx].post[idx_inside] = res;
    if (res > atomicMaxFloat(&post_changes[idx].post[4],res))
    seg[idx] = seg_idx;
*/
    
    int idx_res = threadIdx.x + blockIdx.x*blockDim.x;
    int idx_inside = idx_res%4;
    int idx = idx_res/4;
	if (idx_res>=4*nPts)  return;
    if (border[idx]==0) return;

    int post = post_changes[idx].skip_post[idx_inside];
    int seg_idx = post_changes[idx].changes[idx_inside];
    if(post || ! seg_idx) return;
    float res = -1000; // some large negative number

    float* imgC = img + idx * 3;
    int x = idx % xdim;  
    int y = idx / xdim;  
    //if (idx>154064)
    //    printf("%d ,%d , %d\n",idx_inside, idx, seg_idx);
    const float x0 = imgC[0]-sp_params[seg_idx].mu_i.x;
    const float x1 = imgC[1]-sp_params[seg_idx].mu_i.y;
    const float x2 = imgC[2]-sp_params[seg_idx].mu_i.z;

    const int d0 = x - sp_params[seg_idx].mu_s.x;
    const int d1 = y - sp_params[seg_idx].mu_s.y;
    //color component
    const float J_i_x = J_i.x;
    const float J_i_y = J_i.y;
    const float J_i_z = J_i.z;
    const float sigma_s_x = sp_params[seg_idx].sigma_s.x;
    const float sigma_s_y = sp_params[seg_idx].sigma_s.y;
    const float sigma_s_z = sp_params[seg_idx].sigma_s.z;
    const float logdet_sigma_s = sp_params[seg_idx].logdet_Sigma_s;


    res = res - (x0*x0*J_i_x + x1*x1*J_i_y + x2*x2*J_i_z);   //res = -calc_squared_mahal_3d(imgC,mu_i,J_i);
    res = res -logdet_Sigma_i;
    //space component
    res = res - d0*d0*sigma_s_x;
    res = res - d1*d1*sigma_s_z;
    res = res -  2*d0*d1*sigma_s_y;            // res -= calc_squared_mahal_2d(pt,mu_s,J_s);
    res = res -  logdet_sigma_s;
    if (res > atomicMaxFloat(&post_changes[idx].post[4],res))
    seg[idx] = seg_idx;

    
    
    
    //res += potts_res;

    return;        
}

__global__  void change_seg(int* seg, int* changes, int nPts ,post_changes_helper* post_changes)
{
    
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	if (idx>=nPts)  return;
    //if((!post_changes[idx].changes[0])||(!post_changes[idx].changes[1])||(!post_changes[idx].changes[2])||(!post_changes[idx].changes[3])) return;
    float res_max = -99999;
    int final_idx = seg[idx];
    for (int i=0; i<4 ; i++)
    {
        int idx_change = post_changes[idx].changes[i];
        if(idx_change)
        {
            float val = post_changes[idx].post[i];
            if(res_max < val)
            {
                    res_max = val;
                    final_idx = idx_change;
            }
        }
    }
    seg[idx] = final_idx;
/*
    if (res_max < post_changes[idx].post[0]){
        res_max= post_changes[idx].post[0];
        seg[idx]= post_changes[idx].changes[0];

    }
    if (res_max < post_changes[idx].post[1]){
        res_max= post_changes[idx].post[1];
        seg[idx]= post_changes[idx].changes[1];

    }
    if (res_max < post_changes[idx].post[2]){
    res_max= post_changes[idx].post[2];
    seg[idx]= post_changes[idx].changes[2];

}
    if (res_max < post_changes[idx].post[3]){
    res_max= post_changes[idx].post[3];
    seg[idx]= post_changes[idx].changes[3];

}
    //seg[idx] =arg_max;
*/
    return;        
}




__host__ void update_seg(float* img, int* seg, int* seg_potts_label ,bool* border,
                        superpixel_params* sp_params, 
                        const float3 J_i, const float logdet_Sigma_i, 
                        bool cal_cov, float i_std, int s_std,
                        int nInnerIters,
                        const int nPixels, const int nSPs, int nSPs_buffer, int xdim, int ydim, float beta_potts_term, post_changes_helper* post_changes){
    
    int num_block = ceil( double(nPixels) / double(THREADS_PER_BLOCK) ); 
    int num_block2 = ceil( double(nPixels*4) / double(THREADS_PER_BLOCK) ); 

    dim3 ThreadPerBlock(THREADS_PER_BLOCK,1);
    dim3 BlockPerGrid(num_block,1);
    dim3 BlockPerGrid2(num_block2,1);

	dim3 blockSize(8, 8);
	dim3 gridSize((int)ceil((float)xdim*2 / (float)blockSize.x), (int)ceil((float)ydim*2 / (float)blockSize.y));


    int single_border = 0 ;
    hipMemset(post_changes, 0, nPixels*sizeof(post_changes_helper));
    for (int iter = 0 ; iter < nInnerIters; iter++){
    	// strides of 2*2
        hipMemset(border, 0, nPixels*sizeof(bool));

        find_border_pixels<<<BlockPerGrid,ThreadPerBlock>>>(seg, border, nPixels, xdim, ydim, single_border);

        for (int xmod3 = 0 ; xmod3 <2; xmod3++){
            for (int ymod3 = 0; ymod3 <2; ymod3++){
                //find the border pixels
                //find_border_pixels<<<BlockPerGrid,ThreadPerBlock>>>(seg, border, nPixels, xdim, ydim, single_border);
                //update_seg_subset<<<BlockPerGrid2,ThreadPerBlock>>>(img, seg, seg_potts_label,border, sp_params, J_i, logdet_Sigma_i,  cal_cov, i_std, s_std, nPixels, nSPs,xdim, ydim, xmod3, ymod3, beta_potts_term,post_changes);
                //update_seg_subset<<<gridSize,blockSize>>>(img, seg, seg_potts_label,border, sp_params, J_i, logdet_Sigma_i,  cal_cov, i_std, s_std, nPixels, nSPs,xdim, ydim, xmod3, ymod3, beta_potts_term,post_changes);
                update_seg_subset<<<BlockPerGrid,ThreadPerBlock>>>(img, seg, seg_potts_label,border, sp_params, J_i, logdet_Sigma_i,  cal_cov, i_std, s_std, nPixels, nSPs,xdim, ydim, xmod3, ymod3, beta_potts_term,post_changes);

                //cal_posterior<<<BlockPerGrid2,ThreadPerBlock>>>(img, seg, border, sp_params, J_i, logdet_Sigma_i, i_std,  s_std, 0,  nPixels ,xdim, post_changes);
                //change_seg<<<BlockPerGrid,ThreadPerBlock>>>(seg, 0, nPixels ,post_changes);

            }
        }
    }
    hipMemset(border, 0, nPixels*sizeof(bool));

    find_border_pixels<<<BlockPerGrid,ThreadPerBlock>>>(seg, border, nPixels, xdim, ydim, single_border);

}


/*
* Update the superpixel labels for pixels 
* that are on the boundary of the superpixels
* and on the (xmod3, ymod3) position of 3*3 block
*/
/*
__global__  void update_seg_subset(
    float* img, int* seg, int* seg_potts_label, bool* border,
    superpixel_params* sp_params, 
    const float3 J_i, const float logdet_Sigma_i,  
    bool cal_cov, float i_std, int s_std, 
    const int nPts,const int nSuperpixels,
    const int xdim, const int ydim,
    const int xmod3, const int ymod3, const float beta_potts_term, post_changes_helper* post_changes)
{   



    int idx = threadIdx.x + blockIdx.x*blockDim.x;
   // idx = idx_img;

    if (idx >= 4*nPts)  return;
    int seg_idx = idx/4; 
    int idx_inside = idx%4;
    if (border[seg_idx]==0) return;

    // strides of 2*2
    int x = seg_idx % xdim;  
    if (x % 2 != xmod3) return;
    int y = seg_idx / xdim;   
    if (y % 2 != ymod3) return;
    int idx_cache = threadIdx.x/4;
    float beta = 0;
    //printf("(%d, %d) - %d, %d, %d \n", x,y , idx_cache,threadIdx.x );
    const bool x_greater_than_1 = (x>1);
    const bool y_greater_than_1 = (y>1);
    const bool x_smaller_than_xdim_minus_1 = x<(xdim-1);
    const bool y_smaller_than_ydim_minus_1 = y<(ydim-1);
    if ((!x_greater_than_1)||(!y_greater_than_1)||(!x_smaller_than_xdim_minus_1)||(!y_smaller_than_ydim_minus_1)) return;
    
    __shared__ int N_shared[THREADS_PER_BLOCK/4];
    __shared__ int S_shared[THREADS_PER_BLOCK/4];
    __shared__ int E_shared[THREADS_PER_BLOCK/4];
    __shared__ int W_shared[THREADS_PER_BLOCK/4];
    __shared__ int SW_shared[THREADS_PER_BLOCK/4];
    __shared__ int SE_shared[THREADS_PER_BLOCK/4];
    __shared__ int N_Prev_shared[THREADS_PER_BLOCK/4];
    __shared__ int N_Count[THREADS_PER_BLOCK/4];
    __shared__ int S_Count[THREADS_PER_BLOCK/4];
    __shared__ int E_Count[THREADS_PER_BLOCK/4];
    __shared__ int W_Count[THREADS_PER_BLOCK/4];
    

        //int C = seg[seg_idx]; // center 
    int N,S,E,W, NW, NE , SW, SE; // north, south, east,west        

    // N = S = W = E = OUT_OF_BOUNDS_LABEL; // init to out-of-bounds 
    
    bool nbrs[9];
    //float potts_term[4];
    //potts_term[0] = potts_term[1] = potts_term[2] = potts_term[3] = 0;
    bool isNvalid = 0 ;
    bool isSvalid = 0;
    bool isEvalid = 0;
    bool isWvalid = 0 ; 
    int count_diff_nbrs_N=0;
    int count_diff_nbrs_S=0;
    int count_diff_nbrs_E=0;
    int count_diff_nbrs_W=0;
    if(idx_inside==0)
    {



    //NW =N = NE =W = E = SW = S = SE=5 ;
    // init       

        post_changes[seg_idx].post[4] = -9999;
        NW =__ldg(&seg[seg_idx-xdim-1]);
        N = __ldg(&seg[seg_idx-xdim]);
        NE = __ldg(&seg[seg_idx-xdim+1]);

    }

    if(idx_inside==1)
    {
        W = __ldg(&seg[seg_idx-1]);
        E = __ldg(&seg[seg_idx+1]);
        W_shared[idx_cache] = W;
        E_shared[idx_cache] = E;
    }

    if(idx_inside==2)
    {
        SW = __ldg(&seg[seg_idx+xdim-1]);
        S = __ldg(&seg[seg_idx+xdim]);
        SE =__ldg(&seg[seg_idx+xdim+1]);
        SW_shared[idx_cache] = SW;
        SE_shared[idx_cache] = SE;
        S_shared[idx_cache] = S;
    } 

    __syncthreads();


    if(idx_inside==0)
    {

        W = W_shared[idx_cache] ;
        E = E_shared[idx_cache] ;
        SW = SW_shared[idx_cache] ;
        SE = SE_shared[idx_cache] ;
        S = S_shared[idx_cache];   
        //N :
        set_nbrs(NW, N, NE,  W, E, SW, S, SE,N, nbrs);

        count_diff_nbrs_N = ischangbale_by_nbrs(nbrs);
        isNvalid = nbrs[8];
        //potts_term[0] = calc_potts(beta,count_diff_nbrs_N);

    

        //W :
        set_nbrs(NW, N, NE,  W, E, SW, S, SE,W, nbrs);


        count_diff_nbrs_W = ischangbale_by_nbrs(nbrs);
        isWvalid = nbrs[8];

       // potts_term[1] = calc_potts(beta,count_diff_nbrs_W);

        //S :
        set_nbrs(NW, N, NE,  W, E, SW, S, SE,S, nbrs);


        count_diff_nbrs_S = ischangbale_by_nbrs(nbrs);
        isSvalid = nbrs[8];

        //potts_term[2] = calc_potts(beta,count_diff_nbrs_S);

        //E:
        set_nbrs(NW, N, NE,  W, E, SW, S, SE,E, nbrs);

        // check 8 nbrs and save result if valid to change to the last place of array
        // return how many nbrs different for potts term calculation

        count_diff_nbrs_E = ischangbale_by_nbrs(nbrs);
        isEvalid = nbrs[8];

        //potts_term[3] = calc_potts(beta,count_diff_nbrs_E);

        if(!isNvalid) N = 0;
        if(!isWvalid) W = 0;
        if(!isSvalid) S = 0;
        if(!isEvalid) E = 0;
        




        N_Count[idx_cache] =  count_diff_nbrs_N;
        S_Count[idx_cache] =  count_diff_nbrs_S ;
        E_Count[idx_cache] =  count_diff_nbrs_E;
        W_Count[idx_cache] =  count_diff_nbrs_W;
 
        N_Prev_shared[idx_cache] = (int(isNvalid))+(int(isWvalid))+(int(isSvalid))+(int(isEvalid));
   
    }


    __syncthreads();


    if (N_Prev_shared[idx_cache]<4) return; //BuG for less than 4 
    switch (idx_inside)  
    {
        case (0):
                
        if(!N_shared[idx_cache]) return;
        cal_posterior_new(img,seg,x,y,sp_params,seg_idx,N_shared[idx_cache],J_i,logdet_Sigma_i,i_std,s_std,post_changes,N_Count[idx_cache],beta);
        break;
    
        case (1):

        if(!S_shared[idx_cache]) return;
        if(S_shared[idx_cache]!=N_shared[idx_cache])
        cal_posterior_new(img,seg,x,y,sp_params,seg_idx,S_shared[idx_cache],J_i,logdet_Sigma_i,i_std,s_std,post_changes,S_Count[idx_cache],beta);
        break;


        case (2):

        if(!W_shared[idx_cache]) return;
        if((W_shared[idx_cache]!=S_shared[idx_cache])&&(W_shared[idx_cache]!=N_shared[idx_cache]))        
        cal_posterior_new(img,seg,x,y,sp_params,seg_idx,W_shared[idx_cache],J_i,logdet_Sigma_i,i_std,s_std,post_changes,W_Count[idx_cache],beta);

        case(3):
        if(!E_shared[idx_cache]) return;
        if((E_shared[idx_cache]!=W_shared[idx_cache])&&(E_shared[idx_cache]!=S_shared[idx_cache])&&(E_shared[idx_cache]!=N_shared[idx_cache]))        
        cal_posterior_new(img,seg,x,y,sp_params,seg_idx,E_shared[idx_cache],J_i,logdet_Sigma_i,i_std,s_std,post_changes,E_Count[idx_cache],beta);
        break;
    }
    return;
}

   
*/

__global__  void update_seg_subset(
    float* img, int* seg, int* seg_potts_label, bool* border,
    superpixel_params* sp_params, 
    const float3 J_i, const float logdet_Sigma_i,  
    bool cal_cov, float i_std, int s_std, 
    const int nPts,const int nSuperpixels,
    const int xdim, const int ydim,
    const int xmod3, const int ymod3, const float beta_potts_term, post_changes_helper* post_changes)
{   

    int label_check;
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
   // idx = idx_img;

    int seg_idx = idx; 
    if (seg_idx>=nPts)  return;

    int x = seg_idx % xdim;  
    if (x % 2 != xmod3) return;
    int y = seg_idx / xdim;   
    if (y % 2 != ymod3) return;
    
    if (border[seg_idx]==0) return;
    // strides of 2*2

    //float beta = 4;
    //printf("(%d, %d) - %d, %d, %d \n", x,y , idx_cache,threadIdx.x );
    const bool x_greater_than_1 = (x>1);
    const bool y_greater_than_1 = (y>1);
    const bool x_smaller_than_xdim_minus_1 = x<(xdim-1);
    const bool y_smaller_than_ydim_minus_1 = y<(ydim-1);
    if ((!x_greater_than_1)||(!y_greater_than_1)||(!x_smaller_than_xdim_minus_1)||(!y_smaller_than_ydim_minus_1)) return;
   
   /*if(sp_params[ seg[seg_idx]].count==1) 
    {
        seg[seg_idx]=seg[seg_idx-1];
        return;
    }*/

    




    //int C = seg[seg_idx]; // center 

    // N = S = W = E = OUT_OF_BOUNDS_LABEL; // init to out-of-bounds 
    
    bool nbrs[9];
    //float potts_term[4];
    //potts_term[0] = potts_term[1] = potts_term[2] = potts_term[3] = 0;
    bool isNvalid = 0 ;
    bool isSvalid = 0;
    bool isEvalid = 0;
    bool isWvalid = 0 ; 
    float beta = beta_potts_term;
    //printf("Beta: %f", beta);

    int count_diff_nbrs_N=0;
    int count_diff_nbrs_S=0;
    int count_diff_nbrs_E=0;
    int count_diff_nbrs_W=0;

//NW =N = NE =W = E = SW = S = SE=5 ;
// init       

    float2 res_max;
    res_max.x = -9999;
    //post_changes[seg_idx].post[4] = -9999;
    int NW =__ldg(&seg[seg_idx-xdim-1]);
    int N = __ldg(&seg[seg_idx-xdim]);
    int NE = __ldg(&seg[seg_idx-xdim+1]);
    int W = __ldg(&seg[seg_idx-1]);
    int E = __ldg(&seg[seg_idx+1]);
    int SW = __ldg(&seg[seg_idx+xdim-1]);
    int S = __ldg(&seg[seg_idx+xdim]);
    int SE =__ldg(&seg[seg_idx+xdim+1]);  
    
    //N :
    set_nbrs(NW, N, NE,  W, E, SW, S, SE,N, nbrs);


    count_diff_nbrs_N = ischangbale_by_nbrs(nbrs);
    isNvalid = nbrs[8];
    //potts_term[0] = calc_potts(beta,count_diff_nbrs_N);
     if(!isNvalid) return;
    
    //W :

    set_nbrs(NW, N, NE,  W, E, SW, S, SE,W, nbrs);

    count_diff_nbrs_W = ischangbale_by_nbrs(nbrs);
    isWvalid = nbrs[8];
     if(!isWvalid) return;

    // potts_term[1] = calc_potts(beta,count_diff_nbrs_W);

    //S :
    set_nbrs(NW, N, NE,  W, E, SW, S, SE,S, nbrs);


    count_diff_nbrs_S = ischangbale_by_nbrs(nbrs);
    isSvalid = nbrs[8];
     if(!isSvalid) return;

    //potts_term[2] = calc_potts(beta,count_diff_nbrs_S);

    //E:
    set_nbrs(NW, N, NE,  W, E, SW, S, SE,E, nbrs);

    // check 8 nbrs and save result if valid to change to the last place of array
    // return how many nbrs different for potts term calculation

    count_diff_nbrs_E = ischangbale_by_nbrs(nbrs);
    isEvalid = nbrs[8];
     if(!isEvalid) return;

    //potts_term[3] = calc_potts(beta,count_diff_nbrs_E);






    //N_Prev_shared[idx_cache] = (int(isNvalid))+(int(isWvalid))+(int(isSvalid))+(int(isEvalid));
   
    
    label_check = N;
    res_max =cal_posterior_new(img,seg,x,y,sp_params,seg_idx,label_check,J_i,logdet_Sigma_i,i_std,s_std,post_changes,count_diff_nbrs_N,beta,res_max);

    label_check = S;
    if(label_check!=N)
    res_max = cal_posterior_new(img,seg,x,y,sp_params,seg_idx,label_check,J_i,logdet_Sigma_i,i_std,s_std,post_changes,count_diff_nbrs_S,beta,res_max);


    label_check = W;

    if((label_check!=S)&&(label_check!=N))   
    res_max = cal_posterior_new(img,seg,x,y,sp_params,seg_idx,label_check,J_i,logdet_Sigma_i,i_std,s_std,post_changes,count_diff_nbrs_W,beta,res_max);
    
    label_check = E;

    if((label_check!=W)&&(label_check!=S)&&(label_check!=N))      
    res_max= cal_posterior_new(img,seg,x,y,sp_params,seg_idx,label_check,J_i,logdet_Sigma_i,i_std,s_std,post_changes,count_diff_nbrs_E,beta,res_max);
    
    seg[seg_idx] = res_max.y;
    return;
}

   /*

__global__  void update_seg_subset(
    float* img, int* seg, int* seg_potts_label, bool* border,
    superpixel_params* sp_params, 
    const float3 J_i, const float logdet_Sigma_i,  
    bool cal_cov, float i_std, int s_std, 
    const int nPts,const int nSuperpixels,
    const int xdim, const int ydim,
    const int xmod3, const int ymod3, const float beta_potts_term, post_changes_helper* post_changes)
{   

    int label_check;
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
   // idx = idx_img;

    if (idx >= 4*nPts)  return;
    int seg_idx = idx/4; 
    int idx_inside = idx%4;
    int x = seg_idx % xdim;  
    if (x % 2 != xmod3) return;
    int y = seg_idx / xdim;   
    if (y % 2 != ymod3) return;
    
    if (border[seg_idx]==0) return;

    // strides of 2*2

    int idx_cache = threadIdx.x/4;
    float beta = 0;
    //printf("(%d, %d) - %d, %d, %d \n", x,y , idx_cache,threadIdx.x );
    const bool x_greater_than_1 = (x>1);
    const bool y_greater_than_1 = (y>1);
    const bool x_smaller_than_xdim_minus_1 = x<(xdim-1);
    const bool y_smaller_than_ydim_minus_1 = y<(ydim-1);
    if ((!x_greater_than_1)||(!y_greater_than_1)||(!x_smaller_than_xdim_minus_1)||(!y_smaller_than_ydim_minus_1)) return;
    
    __shared__ int N_shared[THREADS_PER_BLOCK/4];
    __shared__ int S_shared[THREADS_PER_BLOCK/4];
    __shared__ int E_shared[THREADS_PER_BLOCK/4];
    __shared__ int W_shared[THREADS_PER_BLOCK/4];
    __shared__ int N_Prev_shared[THREADS_PER_BLOCK/4];
    __shared__ int N_Count[THREADS_PER_BLOCK/4];
    __shared__ int S_Count[THREADS_PER_BLOCK/4];
    __shared__ int E_Count[THREADS_PER_BLOCK/4];
    __shared__ int W_Count[THREADS_PER_BLOCK/4];
    
    
   
    if(idx_inside==0)
    {

        //int C = seg[seg_idx]; // center 
        int N,S,E,W, NW, NE , SW, SE; // north, south, east,west        

        // N = S = W = E = OUT_OF_BOUNDS_LABEL; // init to out-of-bounds 
        
        bool nbrs[9];
        //float potts_term[4];
        //potts_term[0] = potts_term[1] = potts_term[2] = potts_term[3] = 0;
        bool isNvalid = 0 ;
        bool isSvalid = 0;
        bool isEvalid = 0;
        bool isWvalid = 0 ; 
        float beta = beta_potts_term;
        int count_diff_nbrs_N=0;
        int count_diff_nbrs_S=0;
        int count_diff_nbrs_E=0;
        int count_diff_nbrs_W=0;

    //NW =N = NE =W = E = SW = S = SE=5 ;
    // init       


        post_changes[seg_idx].post[4] = -9999;
        NW =__ldg(&seg[seg_idx-xdim-1]);
        N = __ldg(&seg[seg_idx-xdim]);
        NE = __ldg(&seg[seg_idx-xdim+1]);
        W = __ldg(&seg[seg_idx-1]);
        E = __ldg(&seg[seg_idx+1]);
        SW = __ldg(&seg[seg_idx+xdim-1]);
        S = __ldg(&seg[seg_idx+xdim]);
        SE =__ldg(&seg[seg_idx+xdim+1]);  
      
        //N :
        set_nbrs(NW, N, NE,  W, E, SW, S, SE,N, nbrs);

        count_diff_nbrs_N = ischangbale_by_nbrs(nbrs);
        isNvalid = nbrs[8];
        //potts_term[0] = calc_potts(beta,count_diff_nbrs_N);

        
        //W :
        set_nbrs(NW, N, NE,  W, E, SW, S, SE,W, nbrs);


        count_diff_nbrs_W = ischangbale_by_nbrs(nbrs);
        isWvalid = nbrs[8];

       // potts_term[1] = calc_potts(beta,count_diff_nbrs_W);

        //S :
        set_nbrs(NW, N, NE,  W, E, SW, S, SE,S, nbrs);


        count_diff_nbrs_S = ischangbale_by_nbrs(nbrs);
        isSvalid = nbrs[8];

        //potts_term[2] = calc_potts(beta,count_diff_nbrs_S);

        //E:
        set_nbrs(NW, N, NE,  W, E, SW, S, SE,E, nbrs);

        // check 8 nbrs and save result if valid to change to the last place of array
        // return how many nbrs different for potts term calculation

        count_diff_nbrs_E = ischangbale_by_nbrs(nbrs);
        isEvalid = nbrs[8];

        //potts_term[3] = calc_potts(beta,count_diff_nbrs_E);

        if(!isNvalid) N = 0;
        if(!isWvalid) W = 0;
        if(!isSvalid) S = 0;
        if(!isEvalid) E = 0;
        



        N_shared[idx_cache] = N;
        W_shared[idx_cache] = W;
        E_shared[idx_cache] = E;
        S_shared[idx_cache] = S;
        


        N_Count[idx_cache] =  count_diff_nbrs_N;
        S_Count[idx_cache] =  count_diff_nbrs_S ;
        E_Count[idx_cache] =  count_diff_nbrs_E;
        W_Count[idx_cache] =  count_diff_nbrs_W;
 
        if((!N)||(!W)||(!E)||(!S))  N_Prev_shared[idx_cache] = 1;
        else N_Prev_shared[idx_cache] = 0;
        //N_Prev_shared[idx_cache] = (int(isNvalid))+(int(isWvalid))+(int(isSvalid))+(int(isEvalid));
   
    }


    __syncthreads();


    if (N_Prev_shared[idx_cache]) return; //BuG for less than 4 
    switch (idx_inside)  
    {
        case (0):
        label_check = N_shared[idx_cache];
        if(!label_check) return;
        cal_posterior_new(img,seg,x,y,sp_params,seg_idx,label_check,J_i,logdet_Sigma_i,i_std,s_std,post_changes,N_Count[idx_cache],beta);
        break;
    
        case (1):
        label_check = S_shared[idx_cache];

        if(!label_check) return;
        if(label_check!=N_shared[idx_cache])
        cal_posterior_new(img,seg,x,y,sp_params,seg_idx,label_check,J_i,logdet_Sigma_i,i_std,s_std,post_changes,S_Count[idx_cache],beta);
        break;


        case (2):
        label_check = W_shared[idx_cache];

        if(!label_check) return;
        if((label_check==S_shared[idx_cache])||(label_check==N_shared[idx_cache])) return;    
        cal_posterior_new(img,seg,x,y,sp_params,seg_idx,label_check,J_i,logdet_Sigma_i,i_std,s_std,post_changes,W_Count[idx_cache],beta);
        break;
        
        case(3):
        label_check = E_shared[idx_cache];

        if(!label_check) return;
        if((label_check==W_shared[idx_cache])||(label_check==S_shared[idx_cache])||(label_check==N_shared[idx_cache])) return;        
        cal_posterior_new(img,seg,x,y,sp_params,seg_idx,label_check,J_i,logdet_Sigma_i,i_std,s_std,post_changes,E_Count[idx_cache],beta);
        break;
    }
    return;
}

   
*/
